
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{

  printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main()
{
  printf("GoGo\n");
  hello<<<10, 10>>>();
  // cudaDeviceSynchronize();
  hipError_t cudaerr = hipDeviceSynchronize();
  printf("Done");

  if (cudaerr != hipSuccess)
  {
    printf("kernel launch failed with error \"%s\".\n",
           hipGetErrorString(cudaerr));
  }
}

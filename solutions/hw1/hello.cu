#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void hello()
{

  printf("Hello from block: %u, thread: %u\n", blockidx.x, threadidx.x);
}

int main()
{

  hello<<<2, 2>>>();
  hipDeviceSynchronize();
}
